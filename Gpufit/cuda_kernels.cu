#include "hip/hip_runtime.h"
#include "gpufit.h"
#include "cuda_kernels.cuh"
#include "definitions.h"
#include "linear_1d.cuh"
#include "gauss_1d.cuh"
#include "gauss_2d.cuh"
#include "gauss_2d_elliptic.cuh"
#include "gauss_2d_rotated.cuh"
#include "cauchy_2d_elliptic.cuh"
#include "ramsey_fixed_p.cuh"
#include "ramsey_var_p.cuh"
#include "lse.cuh"
#include "mle.cuh"

/* Description of the cuda_calc_curve_values function
* ===================================================
*
* This function calls one of the fitting curve functions depending on the input
* parameter model_id. The fitting curve function calculates the values of
* the fitting curves and its partial derivatives with respect to the fitting
* curve parameters. Multiple fits are calculated in parallel.
*
* Parameters:
*
* parameters: An input vector of concatenated sets of model parameters.
*
* n_fits: The number of fits.
*
* n_points: The number of data points per fit.
*
* n_parameters: The number of curve parameters.
*
* finished: An input vector which allows the calculation to be skipped for single
*           fits.
*
* values: An output vector of concatenated sets of model function values.
*
* derivatives: An output vector of concatenated sets of model function partial
*              derivatives.
*
* n_fits_per_block: The number of fits calculated by each thread block.
*
* n_blocks_per_fit: The number of thread blocks used to calculate one fit.
*
* model_id: The fitting model ID.
*
* chunk_index: The data chunk index.
*
* user_info: An input vector containing user information.
*
* user_info_size: The size of user_info in bytes.
*
* Calling the cuda_calc_curve_values function
* ===========================================
*
* When calling the function, the blocks and threads must be set up correctly,
* as shown in the following example code.
*
*   dim3  threads(1, 1, 1);
*   dim3  blocks(1, 1, 1);
*
*   threads.x = n_points * n_fits_per_block / n_blocks_per_fit;
*   blocks.x = n_fits / n_fits_per_block * n_blocks_per_fit;
*
*   cuda_calc_curve_values<<< blocks, threads >>>(
*       parameters,
*       n_fits,
*       n_points,
*       n_parameters,
*       finished,
*       values,
*       derivatives,
*       n_fits_per_block,
*       n_blocks_per_fit,
*       model_id,
*       chunk_index,
*       user_info,
*       user_info_size);
*
*/

__global__ void cuda_calc_curve_values(
    float const * parameters,
    int const n_fits,
    int const n_points,
    int const n_parameters,
    int const * finished,
    float * values,
    float * derivatives,
    int const n_fits_per_block,
    int const n_blocks_per_fit,
    int const model_id,
    int const chunk_index,
    char * user_info,
    std::size_t const user_info_size)
{
    int const fit_in_block = threadIdx.x / n_points;
    int const fit_index = blockIdx.x * n_fits_per_block / n_blocks_per_fit + fit_in_block;
    int const fit_piece = blockIdx.x % n_blocks_per_fit;
    int const point_index = threadIdx.x - fit_in_block * n_points + fit_piece * blockDim.x;
    int const first_point = fit_index * n_points;

    float * current_values = values + first_point;
    float * current_derivatives = derivatives + first_point * n_parameters;
    float const * current_parameters = parameters + fit_index * n_parameters;

    if (finished[fit_index])
        return;
    if (point_index >= n_points)
        return;

    if (model_id == GAUSS_1D)
        calculate_gauss1d(current_parameters, n_fits, n_points, current_values, current_derivatives, point_index, fit_index, chunk_index, user_info, user_info_size);
    else if (model_id == GAUSS_2D)
        calculate_gauss2d(current_parameters, n_fits, n_points, current_values, current_derivatives, point_index, fit_index, chunk_index, user_info, user_info_size);
    else if (model_id == GAUSS_2D_ELLIPTIC)
        calculate_gauss2delliptic(current_parameters, n_fits, n_points, current_values, current_derivatives, point_index, fit_index, chunk_index, user_info, user_info_size);
    else if (model_id == GAUSS_2D_ROTATED)
        calculate_gauss2drotated(current_parameters, n_fits, n_points, current_values, current_derivatives, point_index, fit_index, chunk_index, user_info, user_info_size);
    else if (model_id == CAUCHY_2D_ELLIPTIC)
        calculate_cauchy2delliptic(current_parameters, n_fits, n_points, current_values, current_derivatives, point_index, fit_index, chunk_index, user_info, user_info_size);
    else if (model_id == LINEAR_1D)
        calculate_linear1d(current_parameters, n_fits, n_points, current_values, current_derivatives, point_index, fit_index, chunk_index, user_info, user_info_size);
	else if (model_id == RAMSEY_FIXED_P)
        calculate_ramsey_fixed_p(current_parameters, n_fits, n_points, current_values, current_derivatives, point_index, fit_index, chunk_index, user_info, user_info_size);
	else if (model_id == RAMSEY_VAR_P)
        calculate_ramsey_var_p(current_parameters, n_fits, n_points, current_values, current_derivatives, point_index, fit_index, chunk_index, user_info, user_info_size);
}

/* Description of the sum_up_floats function
* ==========================================
*
* This function sums up a vector of float values and stores the result at the
* first place of the vector.
*
* Parameters:
*
* shared_array: An input vector of float values. The vector must be stored
*               on the shared memory of the GPU. The size of this vector must be a
*               power of two. Use zero padding to extend it to the next highest
*               power of 2 greater than the number of elements.
*
* size: The number of elements in the input vector considering zero padding.
*
* Calling the sum_up_floats function
* ==================================
*
* This __device__ function can be only called from a __global__ function or
* an other __device__ function. When calling the function, the blocks and threads
* of the __global__ function must be set up correctly, as shown in the following
* example code.
*
*   dim3  threads(1, 1, 1);
*   dim3  blocks(1, 1, 1);
*
*   threads.x = size * vectors_per_block;
*   blocks.x = n_vectors / vectors_per_block;
*
*   global_function<<< blocks, threads >>>(parameter1, ...);
*
*/

__device__ void sum_up_floats(volatile float* shared_array, int const size)
{
    int const fit_in_block = threadIdx.x / size;
    int const point_index = threadIdx.x - (fit_in_block*size);

    int current_n_points = size >> 1;
    __syncthreads();
    while (current_n_points)
    {
        if (point_index < current_n_points)
        {
            shared_array[point_index] += shared_array[point_index + current_n_points];
        }
        current_n_points >>= 1;
        __syncthreads();
    }
}

/* Description of the cuda_sum_chi_square_subtotals function
* ==========================================================
*
* This function sums up chi_square subtotals in place.
*
* Parameters:
*
* chi_squares: A vector of chi-square values for multiple fits.
*              in: subtotals
*              out: totals
*
* n_blocks_per_fit: The number of blocks used to calculate one fit. It is 
*                   equivalent to the number of subtotals per fit.
*
* n_fits: The number of fits.
*
* finished: An input vector which allows the calculation to be skipped
*           for single fits.
*
* Calling the cuda_sum_chi_square_subtotals function
* ==================================================
*
* When calling the function, the blocks and threads must be set up correctly,
* as shown in the following example code.
*
*   dim3  threads(1, 1, 1);
*   dim3  blocks(1, 1, 1);
*
*   int const example_value = 256;
*
*   threads.x = min(n_fits, example_value);
*   blocks.x = int(ceil(float(n_fits) / float(threads.x)));
*
*   cuda_sum_chi_square_subtotals<<< blocks, threads >>>(
*       chi_squares,
*       n_blocks_per_fit,
*       n_fits,
*       finished);
*
*/

__global__ void cuda_sum_chi_square_subtotals(
    float * chi_squares,
    int const n_blocks_per_fit,
    int const n_fits,
    int const * finished)
{
    int const index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index >= n_fits || finished[index])
        return;

    float * chi_square = chi_squares + index;
    
    double sum = 0.0;
    for (int i = 0; i < n_blocks_per_fit; i++)
        sum += chi_square[i * n_fits];
    
    chi_square[0] = sum;
}

/* Description of the cuda_check_fit_improvement function
* =======================================================
*
* This function checks after each calculation of chi-square values whether the
* currently calculated chi-square values are lower than chi-square values calculated
* in the previous iteration and sets the iteration_failed flags.
*
* Parameters:
*
* iteration_failed: An output vector of flags which indicate whether the fitting
*                   process improved the fit in the last iteration. If yes it is set
*                   to 0 otherwise to 1.
*
* chi_squares: An input vector of chi-square values for multiple fits.
*
* prev_chi_squares: An input vector of chi-square values for multiple fits calculated
*                   in the previous iteration.
*
* n_fits: The number of fits.
*
* finished: An input vector which allows the calculation to be skipped
*           for single fits.
*
* Calling the cuda_check_fit_improvement function
* ===============================================
*
* When calling the function, the blocks and threads must be set up correctly,
* as shown in the following example code.
*
*   dim3  threads(1, 1, 1);
*   dim3  blocks(1, 1, 1);
*
*   int const example_value = 256;
*
*   threads.x = min(n_fits, example_value);
*   blocks.x = int(ceil(float(n_fits) / float(threads.x)));
*
*   cuda_check_fit_improvement <<< blocks, threads >>>(
*       iteration_failed,
*       chi_squares,
*       prev_chi_squares,
*       n_fits,
*       finished);
*
*/

__global__ void cuda_check_fit_improvement(
    int * iteration_failed,
    float const * chi_squares,
    float const * prev_chi_squares,
    int const n_fits,
    int const * finished)
{
    int const index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index >= n_fits || finished[index])
        return;

    bool const prev_chi_squares_initialized = prev_chi_squares[index] != 0.f;
    bool const chi_square_increased = (chi_squares[index] >= prev_chi_squares[index]);
    if (prev_chi_squares_initialized && chi_square_increased)
    {
        iteration_failed[index] = 1;
    }
    else
    {
        iteration_failed[index] = 0;
    }
}

/* Description of the cuda_calculate_chi_squares function
* ========================================================
*
* This function calls one of the estimator funktions depending on the input
* parameter estimator_id. The estimator function calculates the chi-square values.
* The calcluation is performed for multiple fits in parallel.
*
* Parameters:
*
* chi_squares: An output vector of concatenated chi-square values.
*
* states: An output vector of values which indicate whether the fitting process
*         was carreid out correctly or which problem occurred. In this function
*         it is only used for MLE. It is set to 3 if a fitting curve value is
*         negative. This vector includes the states for multiple fits.
*
* data: An input vector of data for multiple fits
*
* values: An input vector of concatenated sets of model function values.
*
* weights: An input vector of values for weighting chi-square, gradient and hessian,
*          while using LSE
*
* n_points: The number of data points per fit.
*
* n_fits: The number of fits.
*
* estimator_id: The estimator ID.
*
* finished: An input vector which allows the calculation to be skipped for single
*           fits.
*
* n_fits_per_block: The number of fits calculated by each thread block.
*
* user_info: An input vector containing user information.
*
* user_info_size: The size of user_info in bytes.
*
* Calling the cuda_calculate_chi_squares function
* ================================================
*
* When calling the function, the blocks and threads must be set up correctly,
* as shown in the following example code.
*
*   dim3  threads(1, 1, 1);
*   dim3  blocks(1, 1, 1);
*
*   threads.x = power_of_two_n_points * n_fits_per_block / n_blocks_per_fit;
*   blocks.x = n_fits / n_fits_per_block * n_blocks_per_fit;
*
*   int const shared_size = sizeof(float) * threads.x;
*
*   cuda_calculate_chi_squares<<< blocks, threads, shared_size >>>(
*       chi_squares,
*       states,
*       data,
*       values,
*       weights,
*       n_points,
*       n_fits,
*       estimator_id,
*       finished,
*       n_fits_per_block,
*       user_info,
*       user_info_size);
*
*/

__global__ void cuda_calculate_chi_squares(
    float * chi_squares,
    int * states,
    float const * data,
    float const * values,
    float const * weights,
    int const n_points,
    int const n_fits,
    int const estimator_id,
    int const * finished,
    int const n_fits_per_block,
    char * user_info,
    std::size_t const user_info_size)
{
    int const shared_size = blockDim.x / n_fits_per_block;
    int const fit_in_block = threadIdx.x / shared_size;
    int const fit_piece = blockIdx.x / n_fits;
    int const fit_index = blockIdx.x * n_fits_per_block + fit_in_block - fit_piece * n_fits;
    int const point_index = threadIdx.x - fit_in_block * shared_size + fit_piece * shared_size;
    int const first_point = fit_index * n_points;

    if (finished[fit_index])
    {
        return;
    }

    float const * current_data = &data[first_point];
    float const * current_weight = weights ? &weights[first_point] : NULL;
    float const * current_value  = &values[first_point];
    int * current_state = &states[fit_index];

    extern __shared__ float extern_array[];
    
    volatile float * shared_chi_square
        = extern_array + (fit_in_block - fit_piece) * shared_size;
    
    if (point_index >= n_points)
    {
        shared_chi_square[point_index] = 0.f;
    }

    if (point_index < n_points)
    {
        if (estimator_id == LSE)
        {
            calculate_chi_square_lse(
                shared_chi_square,
                point_index,
                current_data,
                current_value,
                current_weight,
                current_state,
                user_info,
                user_info_size);
        }
        else if (estimator_id == MLE)
        {
            calculate_chi_square_mle(
                shared_chi_square,
                point_index,
                current_data,
                current_value,
                current_weight,
                current_state,
                user_info,
                user_info_size);
        }
    }
    shared_chi_square += fit_piece * shared_size;
    sum_up_floats(shared_chi_square, shared_size);
    chi_squares[fit_index + fit_piece * n_fits] = shared_chi_square[0];
}

/* Description of the cuda_sum_gradient_subtotals function
* ========================================================
*
* This function sums up the chi-square gradient subtotals in place.
*
* Parameters:
*
* gradients: A vector of gradient values for multiple fits.
*            in: subtotals
*            out: totals
*
* n_blocks_per_fit: The number of blocks used to calculate one fit
*
* n_fits: The number of fits.
*
* n_parameters_to_fit: The number of model parameters, that are not held fixed. 
*
* skip: An input vector which allows the calculation to be skipped for single fits.
*
* finished: An input vector which allows the calculation to be skipped for single
*           fits.
*
* Calling the cuda_sum_gradient_subtotals function
* ================================================
*
* When calling the function, the blocks and threads must be set up correctly,
* as shown in the following example code.
*
*   dim3  threads(1, 1, 1);
*   dim3  blocks(1, 1, 1);
*
*   int const example_value = 256;
*
*   threads.x = min(n_fits, example_value);
*   blocks.x = int(ceil(float(n_fits) / float(threads.x)));
*
*   cuda_sum_gradient_subtotals<<< blocks,threads >>>(
*       gradients,
*       n_blocks_per_fit,
*       n_fits,
*       n_parameters_to_fit,
*       skip,
*       finished);
*
*/

__global__ void cuda_sum_gradient_subtotals(
    float * gradients,
    int const n_blocks_per_fit,
    int const n_fits,
    int const n_parameters,
    int const * skip,
    int const * finished)
{
    int const index = blockIdx.x * blockDim.x + threadIdx.x;
    int const fit_index = index / n_parameters;

    if (fit_index >= n_fits || finished[fit_index] || skip[fit_index])
        return;

    float * gradient = gradients + index;

    double sum = 0.0;
    for (int i = 0; i < n_blocks_per_fit; i++)
        sum += gradient[i * n_fits * n_parameters];

    gradient[0] = sum;
}

/* Description of the cuda_calculate_gradients function
* =====================================================
*
* This function calls one of the gradient functions depending on the input
* parameter estimator_id. The gradient function calculates the gradient values
* of the chi-square function calling a __device__ function. The calcluation is
* performed for multiple fits in parallel.
*
* Parameters:
*
* gradients: An output vector of concatenated sets of gradient vector values.
*
* data: An input vector of data for multiple fits
*
* values: An input vector of concatenated sets of model function values.
*
* derivatives: An input vector of concatenated sets of model function partial
*              derivatives.
*
* weights: An input vector of values for weighting chi-square, gradient and hessian,
*          while using LSE
*
* n_points: The number of data points per fit.
*
* n_fits: The number of fits.
*
* n_parameters: The number of fitting curve parameters.
*
* n_parameters_to_fit: The number of fitting curve parameters, that are not held
*                      fixed.
*
* parameters_to_fit_indices: An input vector of indices of fitting curve parameters,
*                            that are not held fixed.
*
* estimator_id: The estimator ID.
*
* finished: An input vector which allows the calculation to be skipped for single
*           fits.
*
* skip: An input vector which allows the calculation to be skipped for single fits.
*
* n_fits_per_block: The number of fits calculated by each thread block.
*
* user_info: An input vector containing user information.
*
* user_info_size: The number of elements in user_info.
*
* Calling the cuda_calculate_gradients function
* =============================================
*
* When calling the function, the blocks and threads must be set up correctly,
* as shown in the following example code.
*
*   dim3  threads(1, 1, 1);
*   dim3  blocks(1, 1, 1);
*
*   threads.x = power_of_two_n_points * n_fits_per_block / n_blocks_per_fit;
*   blocks.x = n_fits / n_fits_per_block * n_blocks_per_fit;
*
*   int const shared_size = sizeof(float) * threads.x;
*
*   cuda_calculate_gradients<<< blocks, threads, shared_size >>>(
*       gradients,
*       data,
*       values,
*       derivatives,
*       weight,
*       n_points,
*       n_fits,
*       n_parameters,
*       n_parameters_to_fit,
*       parameters_to_fit_indices,
*       estimator_id,
*       finished,
*       skip,
*       n_fits_per_block,
*       user_info,
*       user_info_size);
*
*/

__global__ void cuda_calculate_gradients(
    float * gradients,
    float const * data,
    float const * values,
    float const * derivatives,
    float const * weights,
    int const n_points,
    int const n_fits,
    int const n_parameters,
    int const n_parameters_to_fit,
    int const * parameters_to_fit_indices,
    int const estimator_id,
    int const * finished,
    int const * skip,
    int const n_fits_per_block,
    char * user_info,
    std::size_t const user_info_size)
{
    int const shared_size = blockDim.x / n_fits_per_block;
    int const fit_in_block = threadIdx.x / shared_size;
    int const fit_piece = blockIdx.x / n_fits;
    int const fit_index = blockIdx.x * n_fits_per_block + fit_in_block - fit_piece * n_fits;
    int const point_index = threadIdx.x - fit_in_block * shared_size + fit_piece * shared_size;
    int const first_point = fit_index * n_points;

    if (finished[fit_index] || skip[fit_index])
    {
        return;
    }

    float const * current_data = &data[first_point];
    float const * current_weight = weights ? &weights[first_point] : NULL;
    float const * current_derivative = &derivatives[first_point * n_parameters];
    float const * current_value = &values[first_point];

    extern __shared__ float extern_array[];

    volatile float * shared_gradient = extern_array + (fit_in_block - fit_piece) * shared_size;

    if (point_index >= n_points)
    {
        shared_gradient[point_index] = 0.f;
    }

    for (int parameter_index = 0; parameter_index < n_parameters_to_fit; parameter_index++)
    {
        if (point_index < n_points)
        {
            int const derivative_index  = parameters_to_fit_indices[parameter_index] * n_points + point_index;

            if (estimator_id == LSE)
            {
                calculate_gradient_lse(
                    shared_gradient,
                    point_index,
                    derivative_index,
                    current_data,
                    current_value,
                    current_derivative,
                    current_weight,
                    user_info,
                    user_info_size);
            }
            else if (estimator_id == MLE)
            {
                calculate_gradient_mle(
                    shared_gradient,
                    point_index,
                    derivative_index,
                    current_data,
                    current_value,
                    current_derivative,
                    current_weight,
                    user_info,
                    user_info_size);
            }
        }
        sum_up_floats(shared_gradient + fit_piece * shared_size, shared_size);
        gradients[(fit_index * n_parameters_to_fit + parameter_index) + fit_piece * n_fits * n_parameters_to_fit]
            = shared_gradient[fit_piece * shared_size];
    }
}

/* Description of the cuda_calculate_hessians function
* ====================================================
*
* This function calls one of the hessian function depending on the input
* parameter estimator_id. The hessian funcion calculates the hessian matrix
* values of the chi-square function calling a __device__ functions. The
* calcluation is performed for multiple fits in parallel.
*
* Parameters:
*
* hessians: An output vector of concatenated sets of hessian matrix values.
*
* data: An input vector of data for multiple fits
*
* values: An input vector of concatenated sets of model function values.
*
* derivatives: An input vector of concatenated sets of model function partial
*              derivatives.
*
* weights: An input vector of values for weighting chi-square, gradient and hessian,
*          while using LSE
*
* n_points: The number of data points per fit.
*
* n_parameters: The number of fitting curve parameters.
*
* n_parameters_to_fit: The number of fitting curve parameters, that are not held
*                      fixed.
*
* parameters_to_fit_indices: An input vector of indices of fitting curve parameters,
*                            that are not held fixed.
*
* estimator_id: The estimator ID.
*
* skip: An input vector which allows the calculation to be skipped for single fits.
*
* finished: An input vector which allows the calculation to be skipped for single
*           fits.
*
* user_info: An input vector containing user information.
*
* user_info_size: The size of user_info in bytes.
*
* Calling the cuda_calculate_hessians function
* ============================================
*
* When calling the function, the blocks and threads must be set up correctly,
* as shown in the following example code.
*
*   dim3  threads(1, 1, 1);
*   dim3  blocks(1, 1, 1);
*
*   threads.x = n_parameters_to_fit;
*   threads.y = n_parameters_to_fit;
*   blocks.x = n_fits;
*
*   cuda_calculate_hessians<<< blocks, threads >>>(
*       hessians,
*       data,
*       values,
*       derivatives,
*       weight,
*       n_points,
*       n_parameters,
*       n_parameters_to_fit,
*       parameters_to_fit_indices,
*       estimator_id,
*       skip,
*       finished,
*       user_info,
*       user_info_size);
*
*/

__global__ void cuda_calculate_hessians(
    float * hessians,
    float const * data,
    float const * values,
    float const * derivatives,
    float const * weights,
    int const n_points,
    int const n_parameters,
    int const n_parameters_to_fit,
    int const * parameters_to_fit_indices,
    int const estimator_id,
    int const * skip,
    int const * finished,
    char * user_info,
    std::size_t const user_info_size)
{
    int const fit_index = blockIdx.x;
    int const first_point = fit_index * n_points;

    int const parameter_index_i = threadIdx.x;
    int const parameter_index_j = threadIdx.y;

    if (finished[fit_index] || skip[fit_index])
    {
        return;
    }

    float * current_hessian = &hessians[fit_index * n_parameters_to_fit * n_parameters_to_fit];
    float const * current_data = &data[first_point];
    float const * current_weight = weights ? &weights[first_point] : NULL;
    float const * current_derivative = &derivatives[first_point*n_parameters];
    float const * current_value = &values[first_point];

    int const hessian_index_ij = parameter_index_i * n_parameters_to_fit + parameter_index_j;
    int const derivative_index_i = parameters_to_fit_indices[parameter_index_i] * n_points;
    int const derivative_index_j = parameters_to_fit_indices[parameter_index_j] * n_points;

    double sum = 0.0;
    for (int point_index = 0; point_index < n_points; point_index++)
    {
        if (estimator_id == LSE)
        {
            calculate_hessian_lse(
                &sum,
                point_index,
                derivative_index_i + point_index,
                derivative_index_j + point_index,
                current_data,
                current_value,
                current_derivative,
                current_weight,
                user_info,
                user_info_size);
        }
        else if (estimator_id == MLE)
        {
            calculate_hessian_mle(
                &sum,
                point_index,
                derivative_index_i + point_index,
                derivative_index_j + point_index,
                current_data,
                current_value,
                current_derivative,
                current_weight,
                user_info,
                user_info_size);
        }
    }
    current_hessian[hessian_index_ij] = sum;
}

/* Description of the cuda_modify_step_widths function
* ====================================================
*
* This function midifies the diagonal elements of the hessian matrices by multiplying
* them by the factor (1+ lambda). This operation controls the step widths of the
* iteration. If the last iteration failed, befor modifying the hessian, the diagonal
* elements of the hessian are calculated back to represent unmodified values.
*
* hessians: An input and output vector of hessian matrices, which are modified by
*           the lambda values.
*
* lambdas: An input vector of values for modifying the hessians.
*
* n_parameters: The number of fitting curve parameters.
*
* iteration_failed: An input vector which indicates whether the previous iteration
*                   failed.
*
* finished: An input vector which allows the calculation to be skipped for single fits.
*
* n_fits_per_block: The number of fits calculated by each thread block.
*
* Calling the cuda_modify_step_widths function
* ============================================
*
* When calling the function, the blocks and threads must be set up correctly,
* as shown in the following example code.
*
*   dim3  threads(1, 1, 1);
*   dim3  blocks(1, 1, 1);
*
*   threads.x = n_parameters_to_fit * n_fits_per_block;
*   blocks.x = n_fits / n_fits_per_block;
*
*   cuda_modify_step_width<<< blocks, threads >>>(
*       hessians,
*       lambdas,
*       n_parameters,
*       iteration_failed,
*       finished,
*       n_fits_per_block);
*
*/
        
__global__ void cuda_modify_step_widths(
    float * hessians,
    float const * lambdas,
    unsigned int const n_parameters,
    int const * iteration_failed,
    int const * finished,
    int const n_fits_per_block)
{
    int const shared_size = blockDim.x / n_fits_per_block;
    int const fit_in_block = threadIdx.x / shared_size;
    int const parameter_index = threadIdx.x - fit_in_block * shared_size;
    int const fit_index = blockIdx.x * n_fits_per_block + fit_in_block;

    if (finished[fit_index])
    {
        return;
    }

    float * current_hessian = &hessians[fit_index * n_parameters * n_parameters];

    if (iteration_failed[fit_index])
    {
        current_hessian[parameter_index * n_parameters + parameter_index]
            = current_hessian[parameter_index * n_parameters + parameter_index]
            / (1.0f + lambdas[fit_index] / 10.f);
    }
    
    current_hessian[parameter_index * n_parameters + parameter_index]
        = current_hessian[parameter_index * n_parameters + parameter_index]
        * (1.0f + lambdas[fit_index]);
}

/* Description of the cuda_update_parameters function
* ===================================================
*
* This function stores the fitting curve parameter values in prev_parameters and
* updates them after each iteration.
*
* Parameters:
*
* parameters: An input and output vector of concatenated sets of model
*             parameters.
*
* prev_parameters: An input and output vector of concatenated sets of model
*                  parameters calculated by the previous iteration.
*
* deltas: An input vector of concatenated delta values, which are added to the
*         model parameters.
*
* n_parameters_to_fit: The number of fitted curve parameters.
*
* parameters_to_fit_indices: The indices of fitted curve parameters.
*
* finished: An input vector which allows the parameter update to be skipped for single fits.
*
* n_fits_per_block: The number of fits calculated by each threadblock.
*
* Calling the cuda_update_parameters function
* ===========================================
*
* When calling the function, the blocks and threads must be set up correctly,
* as shown in the following example code.
*
*   dim3  threads(1, 1, 1);
*   dim3  blocks(1, 1, 1);
*
*   threads.x = n_parameters * n_fits_per_block;
*   blocks.x = n_fits / n_fits_per_block;
*
*   cuda_update_parameters<<< blocks, threads >>>(
*       parameters,
*       prev_parameters,
*       deltas,
*       n_parameters_to_fit,
*       parameters_to_fit_indices,
*       finished,
*       n_fits_per_block);
*
*/
    
__global__ void cuda_update_parameters(
    float * parameters,
    float * prev_parameters,
    float const * deltas,
    int const n_parameters_to_fit,
    int const * parameters_to_fit_indices,
    int const * finished,
    int const n_fits_per_block)
{
    int const n_parameters = blockDim.x / n_fits_per_block;
    int const fit_in_block = threadIdx.x / n_parameters;
    int const parameter_index = threadIdx.x - fit_in_block * n_parameters;
    int const fit_index = blockIdx.x * n_fits_per_block + fit_in_block;

    float * current_parameters = &parameters[fit_index * n_parameters];
    float * current_prev_parameters = &prev_parameters[fit_index * n_parameters];

    current_prev_parameters[parameter_index] = current_parameters[parameter_index];

    if (finished[fit_index])
    {
        return;
    }

    if (parameter_index >= n_parameters_to_fit)
    {
        return;
    }

    float const * current_deltas = &deltas[fit_index * n_parameters_to_fit];

    current_parameters[parameters_to_fit_indices[parameter_index]] += current_deltas[parameter_index];
}

/* Description of the cuda_update_state_after_gaussjordan function
* ================================================================
*
* This function interprets the singular flag vector of the Gauss Jordan function
* according to this LM implementation.
*
* Parameters:
*
* n_fits: The number of fits.
*
* singular_checks: An input vector used to report whether a fit is singular.  
*
* states: An output vector of values which indicate whether the fitting process
*         was carreid out correctly or which problem occurred. If a hessian
*         matrix of a fit is singular, it is set to 2.
*
* Calling the cuda_update_state_after_gaussjordan function
* ========================================================
*
* When calling the function, the blocks and threads must be set up correctly,
* as shown in the following example code.
*
*   dim3  threads(1, 1, 1);
*   dim3  blocks(1, 1, 1);
*
*   int const example_value = 256;
*
*   threads.x = min(n_fits, example_value);
*   blocks.x = int(ceil(float(n_fits) / float(threads.x)));
*
*   cuda_update_state_after_gaussjordan<<< blocks, threads >>>(
*       n_fits,
*       singular_checks,
*       states);
*
*/

__global__ void cuda_update_state_after_gaussjordan(
    int const n_fits,
    int const * singular_checks,
    int * states)
{
    int const fit_index = blockIdx.x * blockDim.x + threadIdx.x;

    if (fit_index >= n_fits)
    {
        return;
    }

    if (singular_checks[fit_index] == 1)
    {
        states[fit_index] = STATE_SINGULAR_HESSIAN;
    }

}

/* Description of the cuda_check_for_convergence function
* =======================================================
*
* This function checks after each iteration whether the fits are converged or not.
* It also checks whether the set maximum number of iterations is reached.
*
* Parameters:
*
* finished: An input and output vector which allows the calculation to be skipped
*           for single fits.
*
* tolerance: The tolerance value for the convergence set by user.
*
* states: An output vector of values which indicate whether the fitting process
*         was carreid out correctly or which problem occurred. If the maximum
*         number of iterations is reached without converging, it is set to 1. If
*         the fit converged it keeps its initial value of 0.
*
* chi_squares: An input vector of chi-square values for multiple fits. Used for the
*              convergence check.
*
* prev_chi_squares: An input vector of chi-square values for multiple fits calculated
*                   in the previous iteration. Used for the convergence check.
*
* iteration: The value of the current iteration. It is compared to the value
*            of the maximum number of iteration set by user.
*
* max_n_iterations: The maximum number of iterations set by user.
*
* n_fits: The number of fits.
*
* Calling the cuda_check_for_convergence function
* ===============================================
*
* When calling the function, the blocks and threads must be set up correctly,
* as shown in the following example code.
*
*   dim3  threads(1, 1, 1);
*   dim3  blocks(1, 1, 1);
*
*   int const example_value = 256;
*
*   threads.x = min(n_fits, example_value);
*   blocks.x = int(ceil(float(n_fits) / float(threads.x)));
*
*   cuda_check_for_convergence<<< blocks, threads >>>(
*       finished,
*       tolerance,
*       states,
*       chi_squares,
*       prev_chi_squares,
*       iteration,
*       max_n_iterations,
*       n_fits);
*
*/

__global__ void cuda_check_for_convergence(
    int * finished,
    float const tolerance,
    int * states,
    float const * chi_squares,
    float const * prev_chi_squares,
    int const iteration,
    int const max_n_iterations,
    int const n_fits)
{
    int const fit_index = blockIdx.x * blockDim.x + threadIdx.x;

    if (fit_index >= n_fits)
    {
        return;
    }

    if (finished[fit_index])
    {
        return;
    }

    int const fit_found 
        = abs(chi_squares[fit_index] - prev_chi_squares[fit_index]) 
        < tolerance * fmaxf(1, chi_squares[fit_index]);

    int const max_n_iterations_reached = iteration == max_n_iterations - 1;

    if (fit_found)
    {
        finished[fit_index] = 1;
    }
    else if (max_n_iterations_reached)
    {
        states[fit_index] = STATE_MAX_ITERATION;
    }
}

/* Description of the cuda_evaluate_iteration function
* ====================================================
*
* This function evaluates the current iteration.
*   - It marks a fit as finished if a problem occured.
*   - It saves the needed number of iterations if a fit finished.
*   - It checks if all fits finished
*
* Parameters:
*
* all_finished: An output flag, that indicates whether all fits finished.
*
* n_iterations: An output vector of needed iterations for each fit.
*
* finished: An input and output vector which allows the evaluation to be skipped
*           for single fits
*
* iteration: The values of the current iteration.
*
* states: An input vector of values which indicate whether the fitting process
*         was carreid out correctly or which problem occurred.
*
* n_fits: The number of fits.
*
* Calling the cuda_evaluate_iteration function
* ============================================
*
* When calling the function, the blocks and threads must be set up correctly,
* as shown in the following example code.
*
*   dim3  threads(1, 1, 1);
*   dim3  blocks(1, 1, 1);
*
*   int const example_value = 256;
*
*   threads.x = min(n_fits, example_value);
*   blocks.x = int(ceil(float(n_fits) / float(threads.x)));
*
*   cuda_evaluate_iteration<<< blocks, threads >>>(
*       all_finished,
*       n_iterations,
*       finished,
*       iteration,
*       states,
*       n_fits);
*
*/

__global__ void cuda_evaluate_iteration(
    int * all_finished,
    int * n_iterations,
    int * finished,
    int const iteration,
    int const * states,
    int const n_fits)
{
    int const fit_index = blockIdx.x * blockDim.x + threadIdx.x;

    if (fit_index >= n_fits)
    {
        return;
    }

    if (states[fit_index] != STATE_CONVERGED)
    {
        finished[fit_index] = 1;
    }

    if (finished[fit_index] && n_iterations[fit_index] == 0)
    {
        n_iterations[fit_index] = iteration + 1;
    }

    if (!finished[fit_index])
    {
        * all_finished = 0;
    }
}

/* Description of the cuda_prepare_next_iteration function
* ========================================================
*
* This function prepares the next iteration. It either updates previous
* chi-square values or sets currently calculated chi-square values and
* parameters to values calculated by the previous iteration. This function also
* updates lambda values.
*
* Parameters:
*
* lambdas: An output vector of values which control the step width by modifying
*          the diagonal elements of the hessian matrices.
*
* chi_squares: An input and output vector of chi-square values for multiple fits.
*
* prev_chi_squares: An input and output vector of chi-square values for multiple
*                   fits calculated in the previous iteration.
*
* parameters: An output vector of concatenated sets of model parameters.
*
* prev_parameters: An input vector of concatenated sets of model parameters
*                  calculated in the previous iteration.
*
* n_fits: The number of fits.
*
* n_parameters: The number of fitting curve parameters.
*
* Calling the cuda_prepare_next_iteration function
* ================================================
*
* When calling the function, the blocks and threads must be set up correctly,
* as shown in the following example code.
*
*   dim3  threads(1, 1, 1);
*   dim3  blocks(1, 1, 1);
*
*   int const example_value = 256;
*
*   threads.x = min(n_fits, example_value);
*   blocks.x = int(ceil(float(n_fits) / float(threads.x)));
*
*   cuda_prepare_next_iteration<<< blocks, threads >>>(
*       lambdas,
*       chi_squares,
*       prev_chi_squares,
*       parameters,
*       prev_parameters,
*       n_fits,
*       n_parameters);
*
*/

__global__ void cuda_prepare_next_iteration(
    float * lambdas,
    float * chi_squares,
    float * prev_chi_squares,
    float * parameters,
    float const * prev_parameters,
    int const n_fits,
    int const n_parameters)
{
    int const fit_index = blockIdx.x * blockDim.x + threadIdx.x;
        
    if (fit_index >= n_fits)
    {
        return;
    }

    if (chi_squares[fit_index] < prev_chi_squares[fit_index])
    {
        lambdas[fit_index] *= 0.1f;
        prev_chi_squares[fit_index] = chi_squares[fit_index];
    }
    else
    {
        lambdas[fit_index] *= 10.f;
        chi_squares[fit_index] = prev_chi_squares[fit_index];
        for (int iparameter = 0; iparameter < n_parameters; iparameter++)
        {
            parameters[fit_index * n_parameters + iparameter] = prev_parameters[fit_index * n_parameters + iparameter];
        }
    }
}
